//
// Created by fabian on 28.05.21.
//
#include <hip/hip_runtime.h>

#include "Stream.cuh"
#include "helper.cuh"

IntStream::IntStream(int *timestamp,int *value, size_t size) {
    this->host_timestamp = timestamp;
    this->host_values = value;
    this->size = size;
}


void IntStream::print() {
    printf("IntStream\n");
    printf("t|value\n");
    for (int i = 0; i< this->size;i++) {
        printf("%d|%d \n",this->host_timestamp[i],this->host_values[i]);
    }
    printf("end IntStream\n");
}

void IntStream::free_device(){
    CHECK(hipFree(this->device_timestamp));
    CHECK(hipFree(this->device_values));
}


void IntStream::copy_to_device(){
    int sizeAllocate = this->size * sizeof(int);

    CHECK(hipMalloc((int**)&this->device_timestamp, sizeAllocate));
    CHECK(hipMalloc((int**)&this->device_values, sizeAllocate));
    CHECK(hipMemcpy(this->device_timestamp, this->host_timestamp, sizeAllocate, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(this->device_values, this->host_values, sizeAllocate, hipMemcpyHostToDevice));
}

void IntStream::copy_to_host() {
    int sizeAllocate = this->size * sizeof(int);
    //dest,src
    memset(this->host_values, 0, sizeAllocate);
    memset(this->host_timestamp,  0, sizeAllocate);
    CHECK(hipMemcpy(this->host_values, this->device_values, sizeAllocate, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(this->host_timestamp, this->device_timestamp, sizeAllocate, hipMemcpyDeviceToHost));
}


void UnitStream::print() {
    printf("UnitStream\n");
    printf("t\n");
    for (int i = 0; i< this->size;i++) {
        printf("%d \n",this->host_timestamp[i]);
    }
    printf("end UnitStream\n");
}


UnitStream::UnitStream(int*timestamp,size_t size) {
    this->device_timestamp = timestamp;
    this->size = size;
}


void UnitStream::free_device(){
    CHECK(hipFree(this->device_timestamp));
}

void UnitStream::copy_to_device(){
    int sizeAllocate = this->size * sizeof(int);
    CHECK(hipMalloc((int**)&this->device_timestamp, sizeAllocate));
    CHECK(hipMemcpy(this->device_timestamp, this->host_timestamp, sizeAllocate, hipMemcpyHostToDevice));

}

void UnitStream::copy_to_host() {
    int sizeAllocate = this->size * sizeof(int);
    //dest,src
    memset(this->host_timestamp,  0, sizeAllocate);
    CHECK(hipMemcpy(this->host_timestamp, this->device_timestamp, sizeAllocate, hipMemcpyDeviceToHost));
}