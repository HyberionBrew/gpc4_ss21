//
// Created by fabian on 28.05.21.
//
#include <hip/hip_runtime.h>

#include "Stream.cuh"
#include "helper.cuh"

IntStream::IntStream(int *timestamp,int *value, size_t size) {
    this->host_timestamp = timestamp;
    this->host_values = value;
    this->size = size;
    //TODO! this is per object tf. problems
    this->host_offset = (int *) malloc(size* sizeof(int));
    memset( this->host_offset,0,sizeof(int));

}

//DEVICE ONLY dont use
IntStream::IntStream(bool deviceOnly, size_t size) {
    if (deviceOnly) {
        int sizeAllocate = this->size * sizeof(int);
        this->size = size;
        CHECK(hipMalloc((int**)&this->device_timestamp, sizeAllocate));
        CHECK(hipMalloc((int**)&this->device_values, sizeAllocate));
    }
    else{
        printf("U are using this function wrong, just creates uninitalized stream ONLY on device (i.e. can not be copied back)");
        exit(1);
    }
}

void IntStream::print() {
    printf("IntStream\n");
    printf("t|value\n");
    for (int i = *this->host_offset; i< this->size;i++) {
        printf("%d|%d \n",this->host_timestamp[i],this->host_values[i]);
    }
    printf("end IntStream\n");
}

void IntStream::free_device(){
    CHECK(hipFree(this->device_timestamp));
    CHECK(hipFree(this->device_values));
    CHECK(hipFree(this->device_offset));
    free(this->host_offset);
}

//TODO! implement Staged concurrent copy and execute
//https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#memory-optimizations
// i.e. maybe have a function that doesnt just copy but also performs function?
void IntStream::copy_to_device(){
    int sizeAllocate = this->size * sizeof(int);

    CHECK(hipMalloc((int**)&this->device_timestamp, sizeAllocate));
    CHECK(hipMalloc((int**)&this->device_values, sizeAllocate));
    CHECK(hipMalloc((int**)&this->device_offset, sizeof(int)));
    // Async copying - However excectution of the kernel waits for it to complete! (since default stream 0 is used!)
    // However CPU continues
    CHECK(hipMemcpy(this->device_offset, this->host_offset, sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(this->device_timestamp, this->host_timestamp, sizeAllocate, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(this->device_values, this->host_values, sizeAllocate, hipMemcpyHostToDevice));

}

void IntStream::copy_to_host() {
    int sizeAllocate = this->size * sizeof(int);
    //dest,src
    memset(this->host_values, 0, sizeAllocate);
    memset(this->host_timestamp,  0, sizeAllocate);
    memset(this->host_offset,  0, sizeof(int));
    CHECK(hipMemcpy(this->host_values, this->device_values, sizeAllocate, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(this->host_offset, this->device_offset, sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(this->host_timestamp, this->device_timestamp, sizeAllocate, hipMemcpyDeviceToHost));
    printf("host offset;: %d\n",*this->host_offset);
}


void UnitStream::print() {
    printf("UnitStream\n");
    printf("t\n");
    for (int i = 0; i< this->size;i++) {
        printf("%d \n",this->host_timestamp[i]);
    }
    printf("end UnitStream\n");
}


UnitStream::UnitStream(int*timestamp,size_t size) {
    this->host_timestamp = timestamp;
    this->size = size;
    this->host_offset = (int *) malloc(size* sizeof(int));
    memset( this->host_offset,0,sizeof(int));
}


void UnitStream::free_device(){
    CHECK(hipFree(this->device_timestamp));
    CHECK(hipFree(this->device_offset));
    free(this->host_offset);
}

void UnitStream::copy_to_device(){
    int sizeAllocate = this->size * sizeof(int);
    CHECK(hipMalloc((int**)&this->device_timestamp, sizeAllocate));
    CHECK(hipMalloc((int**)&this->device_offset, sizeof(int)));
    CHECK(hipMemcpy(this->device_timestamp, this->host_timestamp, sizeAllocate, hipMemcpyHostToDevice));

    CHECK(hipMemcpy(this->device_offset, this->host_offset, sizeof(int), hipMemcpyHostToDevice));

}

void UnitStream::copy_to_host() {
    int sizeAllocate = this->size * sizeof(int);
    //dest,src
    memset(this->host_timestamp,  0, sizeAllocate);
    memset(this->host_offset,  0, sizeof(int));
    CHECK(hipMemcpy(this->host_timestamp, this->device_timestamp, sizeAllocate, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(this->host_offset, this->device_offset, sizeof(int), hipMemcpyDeviceToHost));
}