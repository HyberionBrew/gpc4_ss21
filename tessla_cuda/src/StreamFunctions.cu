#include "hip/hip_runtime.h"
//
// Created by fabian on 28.05.21.
//
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "main.cuh"
#include "helper.cuh"
#include "Stream.cuh"
#include "StreamFunctions.cuh"
#include "device_information.cuh"

#define MAX_STREAMS 10
//Memory pointer for the streams
//TODO! not used
// implement pointer passed memory (i.e. not as it is done currently!)
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#dynamic-global-memory-allocation-and-operations
// DISCUSS HOW TO BEST DO THIS
// example https://forums.developer.nvidia.com/t/how-to-allocate-global-dynamic-memory-on-device-from-host/71011/2

__device__ int** streamTable[MAX_STREAMS]; // Per-stream pointer

// https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#numa-best-practices
// ADD stream argument to enable multiple kernels in parallel (10.5. Concurrent Kernel Execution)
// Note:Low Medium Priority: Use signed integers rather than unsigned integers as loop counters.
void time(IntStream *input, IntStream *result,hipStream_t stream){
    //already malloced on host at this time
    //are both streams allocated on the device?

    //TODO! asynchronous copying to the device could be done here!
    // check if already on device! if not copy it to device asynchronously and
    // launch kernels piecewise as in
    // https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#numa-best-practices 10.5

    //choose block sizes
    // spawn threads in increments of 32
    int threads = input->size;
    //shift block size until 1024 than shift block size until maximal? Do we have to schedule twice
    int block_size = 1;
    int blocks = 1;

    //cannot schedule all at once
    // 10.3. Thread and Block Heuristics
    // The number of threads per block should be a multiple of 32 threads
    if (MAX_BLOCKS*MAX_THREADS_PER_BLOCK<threads){
        printf("Cannot schedule the whole stream! TODO! implement iterative scheduling \n");
        //return;
    }

    for (int bs = 32; bs <= MAX_THREADS_PER_BLOCK;bs +=32){
        if (block_size > threads){
            break;
        }
        block_size = bs;
    }
    //TODO! check how many MAX_BLOCKS and
    for (int bl=1; bl <= MAX_BLOCKS*1000; bl++){
        blocks = bl;
        if (bl*block_size > threads){
            break;
        }
    }

    //create kernel memory

    //the pointers are now surely on device
    time_cuda<<<blocks,block_size,0,stream>>>(input->device_timestamp, result->device_timestamp, result->device_values, threads,input->device_offset,result->device_offset);


    //kernel free
    printf("Scheduled time() with <<<%d,%d>>> \n",blocks,block_size);

};


void calcThreadsBlocks(int threads, int *block_size, int*blocks){
    *block_size = 1;
    *blocks = 1;
    if (MAX_BLOCKS*MAX_THREADS_PER_BLOCK<threads){
        printf("Cannot schedule the whole stream! TODO! implement iterative scheduling \n");
        //return;
    }
    //schedule in warp size
    for (int bs = 32; bs <= MAX_THREADS_PER_BLOCK;bs +=32){
        if (*block_size > threads){
            break;
        }
        *block_size = bs;
    }
    //TODO! MAX_BLOCKS?
    // the number of blocks per kernel launch should be in the thousands.
    for (int bl=1; bl <= MAX_BLOCKS*1000; bl++){
        *blocks = bl;
        if (bl* (*block_size) > threads){
            break;
        }
    }

    //TODO! make iterative! see last for hints
    if (*blocks > 1024){
        printf("Many Blocks");
        return;
    }
}

void last(IntStream *inputInt, UnitStream *inputUnit, IntStream *result, hipStream_t stream){
    int threads = (int) inputUnit->size;
    int block_size =1;
    int blocks = 1;
    calcThreadsBlocks(threads,&block_size,&blocks);
    int* block_red;
    hipMalloc((void**)&block_red, sizeof(int)*blocks);
    //TODO! check that no expection is thrown at launch!
    last_cuda<<<blocks,block_size,0,stream>>>(block_red, inputInt->device_timestamp, inputInt->device_values, inputUnit->device_timestamp,result->device_timestamp,result->device_values,inputInt->size, threads);
    int leftBlocks = blocks;
    //TODO! implement and check below functions! for schedulings > 1024 blocks
    /* while(leftBlocks>1024)
        calcThreadsBlocks(leftBlocks,&block_size,&blocks);
        reduce_blocks<<<blocks, block_size, 0, stream>>>(block_red, leftBlocks);
        leftBlocks = blocks;
    };*/
    final_reduce<<<1, block_size, 0, stream>>>(block_red, leftBlocks, result->device_offset);

    hipFree(block_red);
    printf("Scheduled last() with <<<%d,%d>>> \n",blocks,block_size);
    printf("RESULT pointer: %d", result->device_offset);
}

__global__ void final_reduce(int* block_red,int size,int* offset) {
    __shared__ int sdata[1024];
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tid = threadIdx.x;
    if (i < size) {
        sdata[tid] = block_red[i];
        __syncthreads();
        for (unsigned int s = (int)1024 / 2; s > 0; s >>= 1) {
            if (s < size){
                if (tid < s) {
                    if ((i+s+1) > size){
                        sdata[tid] += 0;
                    }
                    else {
                        sdata[tid] += sdata[tid + s];
                    }
                }
            }
            __syncthreads();
        }

        if (i == 0){
            *offset = sdata[0];
            printf("The offset: %d \n",*offset);
        }
    }
}
//reduction example followed from: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
__device__ void count_valid(int * sdata,int * output_timestamp,int* valid, int size, int MaxSize, unsigned int tid, const int i){
    //each thread loads one Element from global to shared memory
    sdata[tid] = 0;

    if (output_timestamp[i] < 0) {
        sdata[tid] = 1;
        //printf("%d ? %d\n",i,output_timestamp[i]);
    }
    __syncthreads();
    for (unsigned int s = (int)size / 2; s > 0; s >>= 1) {
        if (s < size){
            if (tid < s) {
                if ((i+s+1) > MaxSize){
                    sdata[tid] += 0;
                }
                else {
                    sdata[tid] += sdata[tid + s];
                }
            }
        }
        __syncthreads();
    }
    //result to array
    if (tid == 0) *valid=sdata[0];
}

//we should also hand this function the number of invalid input values! -> we have invalid values!
//TODO! check what happens for == and adjust >= or > accordingly (/remove else)
// wikipedia binary search: https://en.wikipedia.org/wiki/Binary_search_algorithm
__global__ void last_cuda(int* block_red, int* input_timestamp, int* input_values,int*unit_stream_timestamps,  int* output_timestamps, int* output_values, int intStreamSize, int size){
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tid = threadIdx.x;
    int local_unit_timestamp = unit_stream_timestamps[i];
    __shared__ int sdata[1024];

   //printf("data %d \n",*(sdata));
    int L = 0;
    int R = intStreamSize-1;
    int m = 0;
    output_timestamps[i] = INT_MIN;
    int out =  INT_MIN;

    if (i<size) {

        while (L<=R) {
            // is this needed? TODO! check and discuss
            //maybe it helps? CHECK!
           //__syncthreads();
            m = (int) (L+R)/2;
            if (input_timestamp[m]<local_unit_timestamp){
                L = m + 1;
                out = input_values[m];
                output_timestamps[i] = unit_stream_timestamps[i];
                //output_values[i] = input_values[m];
            }
            else if (input_timestamp[m]>=local_unit_timestamp){
                R = m -1;
            }
            else{
                // how to handle == ? look up!
                out = input_values[m];
                output_timestamps[i] = unit_stream_timestamps[i];
                break;
            }
        }
        output_values[i] = out;
        //__syncthreads(); //should be unneeded
        block_red[blockIdx.x] = 0; //not really needed
        count_valid(sdata,output_timestamps,&block_red[blockIdx.x], 1024,size,tid,i);
    }

}

// working
__global__ void time_cuda(int* input_timestamp, int* output_timestamps, int* output_values,int size, int*offset, int* resultOffset){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= *offset && i<size){
        output_timestamps[i] = input_timestamp[i];
        output_values[i] = input_timestamp[i];
    }
    if (i == 0) *resultOffset = *offset;
}

__global__ void delay_cuda(int* input_timestamp, int* input_values,int*unit_stream_timestamps,  int* output_timestamps, int* output_values, int intStreamSize, int size){
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    //search for timestamps[i] value in unitEvents. If found:
    // counterTarget = input_values[i]
    // counter = 0
    // while (counter != counterTarget)
    //  counter++;
    //  if unitEvent at counter+
    //      reset //i.e. return
    //  if
    //
}