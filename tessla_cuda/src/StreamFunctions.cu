#include "hip/hip_runtime.h"
//
// Created by fabian on 28.05.21.
//
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "main.cuh"
#include "helper.cuh"
#include "Stream.cuh"
#include "StreamFunctions.cuh"
#include "device_information.cuh"
#include "StreamFunctionHelper.cuh"


#define MAX_STREAMS 10
//Memory pointer for the streams
//TODO! not used
// implement pointer passed memory (i.e. not as it is done currently!)
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#dynamic-global-memory-allocation-and-operations
// DISCUSS HOW TO BEST DO THIS
// example https://forums.developer.nvidia.com/t/how-to-allocate-global-dynamic-memory-on-device-from-host/71011/2

__device__ int** streamTable[MAX_STREAMS]; // Per-stream pointer

int simp_compare(const void *a, const void *b) { // TODO: Delete when no longer needed
    return ( *(int*)a - *(int*)b );
}

void delay(IntStream *s, UnitStream *r, UnitStream*result, hipStream_t stream){
    // Prune IntStream s, mark all events that can't possibly trigger because there's a reset event with value -1
    delay_preliminary_prune(s, r, stream);

    // Allocate arrays for search and set reset-UnitStream as first input
    // New output events in each iteration are bounded by size of r
    int *prevResultsTimestamps = (int*) malloc(r->size * sizeof(int));
    memcpy(prevResultsTimestamps, r->host_timestamp, r->size * sizeof(int));
    UnitStream prevResults(prevResultsTimestamps, r->size);
    prevResults.copy_to_device();
    *prevResults.host_offset = (int) r->size;

    int *tempResultsTimestamps = (int*) malloc(r->size * sizeof(int));
    UnitStream tempResults(tempResultsTimestamps, r->size);
    tempResults.copy_to_device();

    int resultIndex = 0; // TODO: Change?
    int prevResultsCount = r->size; // TODO: Change?
    *result->host_offset = (int) result->size; // TODO: Change?

    // Iteratively search for new output events
    while (prevResultsCount > 0) {
        int threads = prevResultsCount;
        int block_size = 1;
        int blocks = 1;
        calcThreadsBlocks(threads, &block_size, &blocks);

        printf("Scheduled delay() with <<<%d,%d>>>, %i threads \n",blocks,block_size, threads);
        delay_cuda<<<blocks, block_size, 0, stream>>>(s->device_timestamp, s->device_values, prevResults.device_timestamp, tempResults.device_timestamp, threads, s->size, s->device_offset, prevResults.device_offset, tempResults.device_offset, stream);
        tempResults.copy_to_host(); 

        // Merge output events into existing output events
        // Sort tempResults to find actual new events (> -1)
        // TODO: Use parallel sort, parallel merge and count_valid
        qsort(tempResults.host_timestamp, threads, sizeof(int), simp_compare); // TODO: Use parallel sort
        int firstResult = -1;
        for (int i = 0; i < threads; i++) {
            //printf("tempResults.host_timestamp[%i] == %i\n", i, tempResults.host_timestamp[i]);
            if (firstResult == -1 && tempResults.host_timestamp[i] > -1)
                firstResult = i;
            if (tempResults.host_timestamp[i] > 0) {
                // Add tempResults to result. TODO: Change?
                result->host_timestamp[resultIndex] = tempResults.host_timestamp[i];
                *result->host_offset -= 1;
                resultIndex++;
            }
        }

        if (firstResult == -1) {
            prevResultsCount = 0;
            break; // TODO: ?
        }

        // Switch prevResults and tempResults to continue search with newly found timestamps
        prevResultsCount = threads - firstResult;
        UnitStream temp = prevResults;
        prevResults = tempResults;
        tempResults = temp;
        *prevResults.host_offset = prevResults.size - prevResultsCount;
        *tempResults.host_offset = 0;
    }

    // TODO: Sort & prune duplicate result
    qsort(result->host_timestamp, result->size, sizeof(int), simp_compare); // TODO: Use parallel sort
    result->copy_to_device();   // Test copies back from device, but result is only on host right now
    printf("SEARCH DONE, %i Results\n", resultIndex);

    // Cleanup
    prevResults.free_device();
    tempResults.free_device();
    free(prevResultsTimestamps);
    free(tempResultsTimestamps);
}

void delay_preliminary_prune(IntStream *s, UnitStream *r, hipStream_t stream) {
    int threads = (int) s->size;
    int block_size = 1;
    int blocks = 1;
    calcThreadsBlocks(threads,&block_size,&blocks);
    
    printf("Scheduled delay_preliminary_prune() with <<<%d,%d>>>, %i threads \n",blocks,block_size, threads);
    delay_cuda_preliminary_prune<<<blocks, block_size, 0, stream>>>(s->device_timestamp, s->device_values, r->device_timestamp, threads, r->size, s->device_offset, r->device_offset, stream);
}

__global__ void delay_cuda_preliminary_prune(int *inputIntTimestamps, int *inputIntValues, int *resetTimestamps, int size, int resetSize, int *offset, int *resetOffset, hipStream_t stream) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tid = threadIdx.x;
    inputIntTimestamps += *offset;
    inputIntValues += *offset;
    resetTimestamps += *resetOffset;

    int m = lookUpNextElement(resetSize, inputIntTimestamps[i], resetTimestamps);
    if (m > -1 && inputIntTimestamps[i] + inputIntValues[i] > resetTimestamps[m])
        inputIntValues[i] = -1;

}


// binary search
// on failure returns INT_MIN
// returns position of the Element with value x
__device__ int lookUpElement(int size,int searchValue, int * input_timestamp){
    int L = 0;
    int R = size;
    int m = INT_MIN;
    int out = INT_MIN;
    //TODO! APPLY OFFSET DUE TO INVALID WEIGHTS

    while (L<=R) {
        // is this needed? TODO! check and discuss
        //maybe it helps? CHECK!
        __syncthreads();
        m = (int) (L+R)/2;
        if (input_timestamp[m]<searchValue){
            L = m + 1;
        }
        else if (input_timestamp[m]>searchValue){
            R = m -1;
        }
        else{
            out = m;
            break;
        }
    }
    return out;
}

// Binary search looking for next highest timestamp instead of exact match
__device__ int lookUpNextElement(int size, int searchValue, int *timestamps) {
    int L = 0;
    int R = size - 1;
    int m = INT_MIN;
    int out = INT_MIN;
    //TODO! APPLY OFFSET DUE TO INVALID WEIGHTS

    if (timestamps[size-1] > searchValue) {
        while (L<=R) {
            m = (int) (L+R)/2;
            if (timestamps[m] <= searchValue) {
                L = m + 1;
            } else {
                out = m;
                R = m - 1;
            }
        }
    }
    return out;
}


__global__ void delay_cuda(int *inputIntTimestamps, int *inputIntValues, int *resetTimestamps, int *results, int size, int inputSize, int *inputOffset, int *resetOffset, int* resultOffset,hipStream_t stream) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    inputIntTimestamps += *inputOffset;
    inputIntValues += *inputOffset;
    resetTimestamps += *resetOffset;
    results += *resultOffset;

    // For each tempEvent, check if there's a matching (valid) event in IntStream s
    int index = lookUpElement(inputSize, resetTimestamps[i], inputIntTimestamps);
    if (index != INT_MIN && inputIntValues[index] != -1) {
        results[i] = inputIntTimestamps[index] + inputIntValues[index];
    } else {
        results[i] = -1;
    }
}

__device__ void delay_cuda_rec(){

}
// https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#numa-best-practices
// ADD stream argument to enable multiple kernels in parallel (10.5. Concurrent Kernel Execution)
// Note:Low Medium Priority: Use signed integers rather than unsigned integers as loop counters.
void time(IntStream *input, IntStream *result,hipStream_t stream){
    int threads = input->size;
    int block_size = 1;
    int blocks = 1;
    calcThreadsBlocks(threads,&block_size,&blocks);
    //set output stream to input stream size
    if (!result->onDevice) {
        int sizeAllocated = input->size * sizeof(int);
        result->size = input->size;
        result->host_timestamp = (int *) malloc(input->size * sizeof(int));
        result->host_values = (int *) malloc(input->size * sizeof(int));
        memset(result->host_timestamp, 0, sizeAllocated);
        memset(result->host_values, 0, sizeAllocated);
        result->copy_to_device(false);
    }
    time_cuda<<<blocks,block_size,0,stream>>>(input->device_timestamp, result->device_timestamp, result->device_values, threads,input->device_offset,result->device_offset);
    printf("Scheduled time() with <<<%d,%d>>> \n",blocks,block_size);
};




void last(IntStream *inputInt, UnitStream *inputUnit, IntStream *result, hipStream_t stream){
    int threads = (int) inputUnit->size;
    int block_size =1;
    int blocks = 1;
    calcThreadsBlocks(threads,&block_size,&blocks);

    //copy result vector to device
    if (!result->onDevice) {
        //TODO! where do we free this?
        int sizeAllocated = inputUnit->size * sizeof(int);
        result->size = inputUnit->size;
        result->host_timestamp = (int *) malloc(inputUnit->size * sizeof(int));
        result->host_values = (int *) malloc(inputUnit->size * sizeof(int));
        memset(result->host_timestamp, 0, sizeAllocated);
        memset(result->host_values, 0, sizeAllocated);
        result->copy_to_device(false);
    }
    //TODO! check that no expection is thrown at launch!
    last_cuda<<<blocks,block_size,0,stream>>>(inputInt->device_timestamp, inputInt->device_values, inputUnit->device_timestamp,result->device_timestamp,result->device_values,inputInt->size, threads,inputInt->device_offset,inputUnit->device_offset);
    calculate_offset<<<blocks, block_size, 0, stream>>>(result->device_timestamp,result->device_offset, threads);
    printf("Scheduled last() with <<<%d,%d>>> \n",blocks,block_size);
}

//reduction example followed from: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
//calculates the number of non valid timestamps
__global__ void calculate_offset(int* timestamps, int* offset, int size){
    __shared__ int sdata[1024];// each thread loadsone element from global to shared memunsigned 

    int tid = threadIdx.x;
    unsigned int i= blockIdx.x*blockDim.x+ threadIdx.x;
    int block_offset = 0;
    sdata[tid] = 0;

    if (i < size){
         //printf(" timestamp %d \n",*(timestamps+i));
         if (*(timestamps+i) < 0){
            sdata[tid] = 1;
         }
    }
    __syncthreads();


    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    __syncthreads();
    if(tid == 0){ 
        block_offset = sdata[0];
        atomicAdd(offset, block_offset);
    }
    

}
__global__ void last_cuda(int* input_timestamp, int* input_values,int*unit_stream_timestamps,  int* output_timestamps, int* output_values, int intStreamSize, int size, int* offsInt, int* offsUnit){
    
    const int i = threadIdx.x + blockIdx.x * blockDim.x;

    //shift accordingly to offset
    unit_stream_timestamps += *offsUnit;
    input_timestamp += *offsInt;
    input_values += *offsInt;

    size -= *offsUnit;
    intStreamSize -= *offsInt;
    output_timestamps[i] = INT_MIN;
    output_timestamps += *offsUnit;
    output_values += *offsUnit;
    int out =  -1;


    //Search for the timestamp per thread
    int local_unit_timestamp = unit_stream_timestamps[i];
    int L = 0;
    int R = intStreamSize-1;
    int m;
    if (i<size) {

        while (L<=R) {
           //__syncthreads();
            m = (int) (L+R)/2;

            if (input_timestamp[m]<local_unit_timestamp){
                L = m + 1;
                out = input_values[m];

                output_timestamps[i] = unit_stream_timestamps[i];
            }
            else if (input_timestamp[m]>=local_unit_timestamp){
                R = m -1;
            }
            else{
                out = input_values[m];
                output_timestamps[i] = unit_stream_timestamps[i];
                break;
            }
        }
    }
    //all have their respective out values
    //the output_values array has been successfully filled
    //now the threads perform an and reduction starting at 0 going to size
    if (i < size){
        output_values[i] = out;
    }
}

// working
__global__ void time_cuda(int* input_timestamp, int* output_timestamps, int* output_values,int size, int*offset, int* resultOffset){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    input_timestamp += *offset;
    output_timestamps += *offset;
    output_values += *offset;
    if ( i<size-*offset){
        output_timestamps[i] = input_timestamp[i];
        output_values[i] = input_timestamp[i];
    }
    if (i == 0) *resultOffset = *offset;
}


__device__ int merge_path(int *a, int *b, int diag, int a_len, int b_len) {
    // Just using UnitStreams for now
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    int begin = max(0, diag - b_len);               // Start of search window
    int end = min(diag, a_len);                     // End of search window
    int mid;

    // Binary search
    while(begin < end){
    
        mid = (end + begin) / 2;
        int a_val = a[mid];
        int b_val = b[diag - 1 - mid];

        if (a_val < b_val) {
            begin = mid + 1;
        }
        else{
            end = mid;
        }
    }
    return begin;
}

// Device internal sequential merge of small partitions
__device__ void merge_serial(int *a, int *b, int *c,
                             int a_start, int b_start,
                             int vpt, int tidx,
                             int a_len, int b_len){

    int a_i = a_start;
    int b_i = b_start;
    int a_val = a[a_i];
    int b_val = b[b_i];
    int size = vpt;

    bool a_done = a_i >= a_len ? true : false;
    bool b_done = b_i >= b_len ? true : false;

    // Could possibly be optimized since only the last block needs range checks
    // #pragma unroll is also an option according to https://moderngpu.github.io/merge.html
    for(int i = 0; i < vpt; ++i) {

        // Break if last block doesn't fit
        if (a_done && b_done){
            break;
        }

        if (a_done){
            c[tidx*vpt + i] = b_val;
            b_i++;
        }
        else if (b_done){
            c[tidx*vpt + i] = a_val;
            a_i++;
        }
        else if (a_val <= b_val){
            c[tidx*vpt + i] = a_val;
            a_i++;
            if (a_val == b_val && (b_i > b_start || tidx == 0)){
                b[b_i] = -1;
            }
           
        }
        else{
            c[tidx*vpt + i] = b_val;
            b_i++;
        }

        if (a_i >= a_len){
            a_done = true;
        }
        else{
            a_val = a[a_i];
        }

        if (b_i >= b_len){
            b_done = true;
        }
        else{
            b_val = b[b_i];
        }
    }

    __syncthreads();

    if (tidx == 0){
        // Thread 0 does not have to check its starting values
        return;
    }

    // Afterwards, threads have to check for overlapping timestamps in their c[] partition!
    // VPT > 1 check not really necessary, we should guarantee that VPT > 1 beforehand, otherwise the mergepath is the full merge anyway
    if (vpt > 1){
        for (int i = 0; i < vpt; i++){
            if (c[tidx*vpt + i - 1] == c[tidx*vpt + i]){
                c[tidx*vpt + i] = -1;
            }
        }
    }
}

// https://stackoverflow.com/questions/30729106/merge-sort-using-cuda-efficient-implementation-for-small-input-arrays
/**
 * See the following paper for parallel merging of sorted arrays:
 * O. Green, R. Mccoll, and D. Bader
 * GPU merge path: a GPU merging algorithm
 * International Conference on Supercomputing
 * November 2014
 * URL: https://www.researchgate.net/publication/254462662_GPU_merge_path_a_GPU_merging_algorithm
 *
 * The paper claims a runtime complexity of O(log n + n/p), p ... # of processors
 */
void merge(UnitStream *s1, UnitStream *s2, UnitStream *result, int threads){

    int block_size = 1;
    int blocks = 1;

    // Export to function maybe -> reusable (e.g. with a config struct)
    if (MAX_BLOCKS*MAX_THREADS_PER_BLOCK<threads){
        printf("Cannot schedule the whole stream! TODO! implement iterative scheduling \n");
        //return;
    }

    for (int bs = 32; bs <= MAX_THREADS_PER_BLOCK;bs +=32){
        if (block_size > threads){
            break;
        }
        block_size = bs;
    }

    for (int bl=1; bl <= MAX_BLOCKS*1000; bl++){
        blocks = bl;
        if (bl*block_size > threads){
            break;
        }
    }

    threads = blocks*block_size;

    // Using the pseudo-code in the paper
    int a_len = sizeof(s1->host_timestamp) / sizeof(s1->host_timestamp[0]);
    int b_len = sizeof(s2->host_timestamp) / sizeof(s2->host_timestamp[0]);
    memset(result->host_timestamp, -1, (a_len + b_len) * sizeof(int));

    // hipMalloc Timestamp arrays
    s1->copy_to_device();
    s2->copy_to_device();
    result->copy_to_device();

    int sha_memsize = (a_len + b_len) * sizeof(int);

    // 3, 2, 1, go
    merge_cuda<<<blocks, block_size, sha_memsize>>>(s1->device_timestamp, s2->device_timestamp, result->device_timestamp, threads, s1->size, s2->size);

    // Copy back results
    result->copy_to_host();
    
    printf("After Merge\n");
    printf("S1: -----------------------------------------\n");
    for (int i = 0; i < s1->size; i++){
        printf("%i, ", s1->host_timestamp[i]);
    }
    printf("\n-----------------------------------------------\n");

    printf("S2: -----------------------------------------\n");
    for (int i = 0; i < s2->size; i++){
        printf("%i, ", s2->host_timestamp[i]);
    }
    printf("\n-----------------------------------------------\n");

    printf("Result: -----------------------------------------\n");
    for (int i = 0; i < result->size; i++){
        printf("%i, ", result->host_timestamp[i]);
    }
    printf("\n-----------------------------------------------\n");
    
}

// https://moderngpu.github.io/merge.html
// https://github.com/moderngpu/moderngpu/blob/V1.1/include/device/ctamerge.cuh
__global__ void merge_cuda(int *a, int *b, int *c, int threads, int a_len, int b_len){

    const int i = threadIdx.x + blockIdx.x * blockDim.x;        // Thread ID

    int vpt = ceil((double)(a_len + b_len) / (double)threads);  // Values per thread
    int diag = i * vpt;                                         // Binary search constraint

    int intersect = merge_path(a, b, diag, a_len, b_len);
    int a_start = intersect;
    int b_start = diag - intersect;

    merge_serial(a, b, c, a_start, b_start, vpt, i, a_len, b_len);
}




