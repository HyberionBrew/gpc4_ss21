#include "hip/hip_runtime.h"
//
// Created by fabian on 28.05.21.
//
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "main.cuh"
#include "helper.cuh"
#include "Stream.cuh"
#include "StreamFunctions.cuh"
#include "device_information.cuh"
// https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#numa-best-practices
// ADD stream argument to enable multiple kernels in parallel (10.5. Concurrent Kernel Execution)
// Note:Low Medium Priority: Use signed integers rather than unsigned integers as loop counters.
void time(IntStream *input, IntStream *result,hipStream_t stream){
    //already malloced on host at this time
    //are both streams allocated on the device?

    //TODO! asynchronous copying to the device could be done here!
    // check if already on device! if not copy it to device asynchronously and
    // launch kernels piecewise as in
    // https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#numa-best-practices 10.5

    //choose block sizes
    // spawn threads in increments of 32
    int threads = input->size;
    //shift block size until 1024 than shift block size until maximal? Do we have to schedule twice
    int block_size = 1;
    int blocks = 1;

    //cannot schedule all at once
    // 10.3. Thread and Block Heuristics
    // The number of threads per block should be a multiple of 32 threads
    if (MAX_BLOCKS*MAX_THREADS_PER_BLOCK<threads){
        printf("Cannot schedule the whole stream! TODO! implement iterative scheduling \n");
        //return;
    }

    for (int bs = 32; bs <= MAX_THREADS_PER_BLOCK;bs +=32){
        if (block_size > threads){
            break;
        }
        block_size = bs;
    }
    //TODO! MAX_BLOCKS?
    // the number of blocks per kernel launch should be in the thousands.
    for (int bl=1; bl <= MAX_BLOCKS*1000; bl++){
        blocks = bl;
        if (bl*block_size > threads){
            break;
        }
    }

    //the pointers are now surely on device
    time_cuda<<<blocks,block_size,0,stream>>>(input->device_timestamp, result->device_timestamp, result->device_values, threads);
    printf("Scheduled with <<<%d,%d>>> \n",blocks,block_size);

};

__global__ void time_cuda(int* input_timestamp, int* output_timestamps, int* output_values,int size){
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<size){
        output_timestamps[i] = input_timestamp[i];
        output_values[i] = input_timestamp[i];
    }
}