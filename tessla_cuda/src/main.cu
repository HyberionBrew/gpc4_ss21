#include "hip/hip_runtime.h"
// The boilerplatte code is taken from tuwel

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include "main.cuh"
#include "helper.cuh"
#include "Stream.cuh"
#include "StreamFunctions.cuh"

void experimental_time(){

}


void experimental_last(){

}


int main(int argc, char **argv) {

    printf("%s Starting...\n", argv[0]);
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    //might wanna derive MAX_THREADS and so on from here! TODO!
    printf("Using Device %d: %s\n", dev, deviceProp.name);

    //create & allocate experimental streams
    //still working for size = 1024*1024*10
    int size = 10000;

    int sizeAllocated = (size_t)size * sizeof(int);
    int * host_timestamp = (int *) malloc(size * sizeof(int));
    int * host_unit_timestamp = (int *) malloc(size * sizeof(int));
    int * host_value = (int *) malloc(size* sizeof(int));
    //int * host_timestamp2 = (int *) malloc(size * sizeof(int));
    //int * host_value2 = (int *) malloc(size* sizeof(int));



    for (int i = 0; i< size;i++) {
        *(host_timestamp+i) = i;
        *(host_unit_timestamp+i) = i;
        *(host_value+i) = i;
    }

    /**(host_timestamp) = 3;
    *(host_timestamp+1) = 6;
    *(host_timestamp+2) = 8;
    *(host_value) = 1;
    *(host_value+1) = 3;
    *(host_value+2) = 6;*/


    *(host_unit_timestamp) = 0;
    *(host_unit_timestamp+1) = 2;
    *(host_unit_timestamp+2) = 4;
    *(host_unit_timestamp+3) = 5;
    *(host_unit_timestamp+4) = 9;

    //initially empty stream values
    int * host_timestampOut = (int *) malloc(size * sizeof(int));
    int * host_valueOut = (int *) malloc(size* sizeof(int));
    int * host_timestampOut2 = (int *) malloc(size * sizeof(int));
    int * host_valueOut2 = (int *) malloc(size* sizeof(int));
    CHECK(hipProfilerStart());

    memset(host_timestampOut2,0,sizeAllocated);
    memset(host_valueOut2,0,sizeAllocated);
    memset(host_timestampOut,0,sizeAllocated);
    memset(host_valueOut,0,sizeAllocated);

    IntStream inputStream(host_timestamp,host_value, size);
    IntStream outputStream(host_timestampOut,host_valueOut,size);
    IntStream outputStream2(host_timestampOut2,host_valueOut2,size);
    //IntStream outputStream2(host_timestampOut2,host_valueOut2,size);
    UnitStream inputUnitStream(host_unit_timestamp,size);
    // create streams for parallel kernel launches
    int MAX_STREAMS = 16; // check if this is really max
    // I think we can +2 streams for in/out sync? but not sure

    // IMPORTANT! NO CONCURRENCY BETWEEN KERNELS POSSIBLE if:
    // 3.2.6.5.4. Implicit Synchronization https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
    //a page-locked host memory allocation,
    //a device memory allocation,
    //a device memory set,
    //a memory copy between two addresses to the same device memory,
    //any CUDA command to the NULL stream,
    //        a switch between the L1/shared memory configurations described in Compute Capability 3.x and Compute Capability 7.x.

    hipStream_t stream[MAX_STREAMS];
    for (int i = 0; i < MAX_STREAMS; ++i)
        hipStreamCreate(&stream[i]);

    //end config
    inputStream.copy_to_device();
    //inputStream.print();
    inputUnitStream.copy_to_device();
    outputStream.copy_to_device();
    //outputStream.copy_to_device();
    //inputStream2.copy_to_device();
    outputStream2.copy_to_device();
    //time(&inputStream, &outputStream, stream[0]);
    //inputUnitStream.print();
    //inputStream.print();
    last(&inputStream, &inputUnitStream, &outputStream, stream[0]);
    time(&outputStream,&outputStream2, stream[0]);
    //copy back and output
    //printf("time \n");
    outputStream2.copy_to_host();
    //outputStream2.print();

    outputStream.copy_to_host();
    //outputStream.print();



    //inputStream2.free_device();
    outputStream.free_device();
    outputStream2.free_device();
    inputStream.free_device();
    inputUnitStream.free_device();

    //free(host_timestampOut2);
    //free(host_valueOut2);
    //free(host_value2);
    //free(host_timestamp2);
    free(host_unit_timestamp);
    free(host_timestampOut);
    free(host_valueOut);
    free(host_value);
    free(host_timestamp);

    for (int i = 0; i < MAX_STREAMS; ++i)
        hipStreamDestroy(stream[i]);
    //not working with destructor! --> I think we just shouldn't use destructor
    CHECK(hipProfilerStop());
    CHECK(hipDeviceReset());


    return(0);
}
