#include "hip/hip_runtime.h"
#include "StreamFunctions.cuh"
#include "StreamFunctionsThrust.cuh"
#include "helper.cuh"
#include "Stream.cuh"
#include <thrust/device_ptr.h>
#include <iostream>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/unique.h>
#include <thrust/merge.h>
#include <thrust/functional.h>


struct is_larger_zero
{
  __host__ __device__
  bool operator()(const int x)
  {
    return x > -1;
  }
};

std::shared_ptr<GPUIntStream> last_thrust(std::shared_ptr<GPUIntStream> inputInt, std::shared_ptr<GPUUnitStream>  inputUnit, hipStream_t stream){
    
    //first cast device pointers to thrust pointers
    auto offsetInt = thrust::device_pointer_cast(inputInt->device_offset);
    auto offsetUnit = thrust::device_pointer_cast(inputUnit->device_offset);
    //shift for offset
    auto inputInt_timestamps = thrust::device_pointer_cast(inputInt->device_timestamp+*offsetInt);
    auto inputInt_values = thrust::device_pointer_cast(inputInt->device_values+*offsetInt);
    auto inputUnit_timestamps = thrust::device_pointer_cast(inputUnit->device_timestamp+*offsetUnit);
    //Standard guard

    std::shared_ptr<GPUIntStream> result = std::make_shared<GPUIntStream>();
    int sizeAllocated = inputUnit->size * sizeof(int);
    result->size = inputUnit->size;
    result->host_timestamp = (int *) malloc(inputUnit->size * sizeof(int));
    result->host_values = (int *) malloc(inputUnit->size * sizeof(int));

    // Check if we have enough memory left
    if (result->host_values == nullptr || result->host_timestamp == nullptr) {
        throw std::runtime_error("Out of memory.");
    }

    memset(result->host_timestamp, 0, sizeAllocated);
    memset(result->host_values, 0, sizeAllocated);
    result->copy_to_device(false);



    auto result_values = thrust::device_pointer_cast(result->device_values);
    auto result_timestamps = thrust::device_pointer_cast(result->device_timestamp);
    auto result_offs = thrust::device_pointer_cast(result->device_offset);
    
    //fill those that are not part of the current calc (since they are invalid) with -1
    thrust::fill(result_values,result_values+*offsetUnit,-1);
    thrust::fill(result_timestamps,result_timestamps+*offsetUnit,-1);
    
    //now only look at valid region
    result_values = thrust::device_pointer_cast(result->device_values+*offsetUnit);
    result_timestamps = thrust::device_pointer_cast(result->device_timestamp+*offsetUnit);
    
    //Actual algorithm starts here!
    thrust::lower_bound(inputInt_timestamps, inputInt_timestamps+inputInt->size-*offsetInt,
                    inputUnit_timestamps, inputUnit_timestamps+inputUnit->size-*offsetUnit, 
                    result_timestamps,
                    thrust::less<int>());
    //decrement by -1
    thrust::transform(result_timestamps,
                  result_timestamps+result->size-*offsetUnit,
                  thrust::make_constant_iterator((1)),
                  result_timestamps,
                  thrust::minus<int>());
    
    //calculate new additional offset
    *result_offs = thrust::count(result_timestamps, result_timestamps+result->size-*offsetUnit, -1);
    
    thrust::gather(result_timestamps,result_timestamps+result->size-*offsetUnit,
                    inputInt_values,
                    result_values);

    //USE COPY_N ! otherwise unsafe
    thrust::copy_n(inputUnit_timestamps+ *result_offs, result->size-*result_offs-*offsetUnit, 
                    result_timestamps+*result_offs);
    
    //final offset calculation
    *result_offs = *result_offs+*offsetUnit;
    return result;
}


struct is_smaller
{

  int _thresh;
    is_smaller(int thresh) : _thresh(thresh) { }

  __host__ __device__
  const bool operator()(const int &x) 
  {
    return x <= _thresh;
  }
};

thrust::device_vector<int> cross_streams(thrust::device_ptr<int> inputInt1_timestamps, thrust::device_ptr<int> inputInt2_timestamps,thrust::device_ptr<int> inputInt1_values,thrust::device_ptr<int> inputInt2_values,int size1,int size2){
    //*inputInt1_timestamps
    int input_shift =0;



    if (size1 > 0){
      // count the odd elements in vec
      int result = thrust::count_if(inputInt2_timestamps, inputInt2_timestamps+size2, is_smaller(*inputInt1_timestamps));
      
      //lower 0 guard
      input_shift = std::max(0,result-1);
      printf("inputshift: %d \n",input_shift);
    }
    else{
      input_shift = 0;
    }
    thrust::device_vector<int> fit_1(size2-input_shift);
    thrust::lower_bound(inputInt1_timestamps, inputInt1_timestamps+size1,
                      inputInt2_timestamps+input_shift, inputInt2_timestamps+size2,
                      fit_1.begin(),
                      thrust::less_equal<int>());
        //decrement by -1
    thrust::transform(fit_1.begin(),
                  fit_1.end(),
                  thrust::make_constant_iterator((1)),
                  fit_1.begin(),
                  thrust::minus<int>());
    
    //calculate new additional offset
    //DEPRECATED
    int invalids = thrust::count(fit_1.begin(), fit_1.end(), -1);
    assert(invalids == 0);

    for(int i = 0; i < fit_1.size(); i++) {
       std::cout << "Fit[" << i << "] = " << fit_1[i] << std::endl;
    }

    printf("size_1: %d \n",size1);
    printf("size_2: %d \n",size2);
    int size2_new = size2-input_shift;
     int out_of_range = thrust::count(fit_1.begin(), fit_1.end(), size1);
    thrust::fill(fit_1.end()-out_of_range,fit_1.end(),size1-1);
    thrust::device_vector<int> added_1(size2_new);
    thrust::gather(fit_1.begin(),fit_1.end(),
                  inputInt1_values,
                  added_1.begin());

    
    for(int i = 0; i < added_1.size(); i++) {
       std::cout << "ADD[" << i << "] = " << added_1[i] << std::endl;
    }

   if (size2 <= size1){
        //first add up for all valid values
        printf("taken\n");
        //count values that would be outside of range, i.e. all values == size
       
        printf("%d \n", out_of_range);
        thrust::transform(added_1.begin(), added_1.end(), inputInt2_values+input_shift,added_1.begin(), thrust::plus<int>());
        //thrust::transform(added_1.end()-out_of_range, added_1.end(), thrust::make_constant_iterator(*(inputInt2_values+size2)),added_1.end()-out_of_range, thrust::plus<int>());
        //now fill up remaining
        //thrust::transform(fit_1[size_fit1], fit_2,fit_2 +size_fit1, op);
        //inputInt2_values +result_values
    }
    else{
      //size_inputInt1 > size_inputIn2
      thrust::transform(added_1.begin(), added_1.end(), inputInt2_values+input_shift,added_1.begin(), thrust::plus<int>());
    }
 
    for(int i = 0; i < added_1.size(); i++) {
        std::cout << "ADDFInal[" << i << "] = " << added_1[i] << std::endl;
    }
  return added_1;
}

typedef thrust::tuple<int, int> tuple_t;
struct tupleEqual
{
  __host__ __device__
    bool operator()(tuple_t x, tuple_t y)
    {
      return ( (x.get<0>()== y.get<0>()) && (x.get<1>() == y.get<1>()) );
    }
};
//TODO! only supports adds
std::shared_ptr<GPUIntStream> slift_thrust(std::shared_ptr<GPUIntStream> inputInt1, std::shared_ptr<GPUIntStream> inputInt2,hipStream_t stream){
    /*PREAMBLE*/
    auto offsetInt1 = thrust::device_pointer_cast(inputInt1->device_offset);
    auto offsetInt2 = thrust::device_pointer_cast(inputInt2->device_offset);
    //shift for offset
    auto inputInt1_timestamps = thrust::device_pointer_cast(inputInt1->device_timestamp+*offsetInt1);
    auto inputInt1_values = thrust::device_pointer_cast(inputInt1->device_values+*offsetInt1);
    auto inputInt2_timestamps = thrust::device_pointer_cast(inputInt2->device_timestamp+*offsetInt2);
    auto inputInt2_values = thrust::device_pointer_cast(inputInt2->device_values+*offsetInt2);
    //Standard guard
    //TODO! but in function @ cleanup!
    std::shared_ptr<GPUIntStream> result = std::make_shared<GPUIntStream>();
    int sizeAllocated = (inputInt1->size+inputInt2->size)*sizeof(int);
    result->size = inputInt1->size+ inputInt2->size;
    result->host_timestamp = (int *) malloc(result->size * sizeof(int));
    result->host_values = (int *) malloc(result->size * sizeof(int));
    memset(result->host_timestamp, 0, sizeAllocated);
    memset(result->host_values, 0, sizeAllocated);
    result->copy_to_device(false);

    auto result_values = thrust::device_pointer_cast(result->device_values);
    auto result_timestamps = thrust::device_pointer_cast(result->device_timestamp);
    auto result_offs = thrust::device_pointer_cast(result->device_offset);
    *result_offs = *offsetInt1 + *offsetInt2;
    //fill those that are not part of the current calc (since they are invalid) with -1
    thrust::fill(result_values,result_values+*result_offs,-1);
    thrust::fill(result_timestamps,result_timestamps+*result_offs,-1);

    //now only look at valid region
    result_values = thrust::device_pointer_cast(result->device_values+*result_offs);
    result_timestamps = thrust::device_pointer_cast(result->device_timestamp+*result_offs);

    /*FINISHED PREAMBLE*/

    //add one stream up

    //calc lower bounds
    int size_inputInt2 = inputInt2->size-*offsetInt2;
    int size_inputInt1 = inputInt1->size-*offsetInt1;
        for(int i = 0; i < size_inputInt2; i++) {
        std::cout << "time[" << i << "] = " << *(inputInt2_timestamps+i) << std::endl;
    }
    thrust::device_vector<int> add = cross_streams(inputInt1_timestamps, inputInt2_timestamps,inputInt1_values,inputInt2_values, size_inputInt1,size_inputInt2);
    for(int i = 0; i < size_inputInt2; i++) {
        std::cout << "time[" << i << "] = " << *(inputInt2_timestamps+i) << std::endl;
    }
    int shift_timestamps1 = size_inputInt2-add.size();
    
    printf("-------------");
    //now shift timestamps accordingly
    thrust::device_vector<int> add2 = cross_streams(inputInt2_timestamps, inputInt1_timestamps,inputInt2_values,inputInt1_values, size_inputInt2,size_inputInt1);
    int shift_timestamps = size_inputInt1-add2.size();
    //inputInt2_timestamps += shift_timestamps1;
    //inputInt1_timestamps += shift_timestamps;
    //print both streams for debugging
     std::cout << "Stream1:" << add.size() << std::endl;
    for (int i = 0; i<add.size();i++){
      std::cout << "S[" << i << "] = "<<*(inputInt2_timestamps+i+shift_timestamps1)<< "|"  << add[i]  << std::endl;
    }

    std::cout << "Stream2:" << std::endl;
    for (int i = 0; i<add2.size();i++){
      std::cout << "S[" << i << "] = " <<*(inputInt1_timestamps+i+shift_timestamps)<< "|"  << add2[i] << std::endl;
    }
    //now shift timestamps accordingly
    //merge into new vector array
    printf("==???");
    thrust::device_vector<int> merged_timestamps(add.size()+add2.size());
    printf("%d \n",merged_timestamps.size());
    thrust::device_vector<int> merged_values(add.size()+add2.size());
    thrust::merge_by_key(inputInt1_timestamps+shift_timestamps,inputInt1_timestamps+size_inputInt1,
                        inputInt2_timestamps+shift_timestamps1,inputInt2_timestamps+size_inputInt2,
                        add2.begin(), add.begin(),merged_timestamps.begin(),merged_values.begin());
  printf("%d \n",merged_timestamps.size());
    for (int i=0 ; i < merged_timestamps.size();i++){
      std::cout << "Final[" << i << "] = " <<merged_timestamps[i]<< std::endl;
    }
    thrust::device_vector<int> timestamps_res(add.size()+add2.size());
    thrust::device_vector<int> values_res(add.size()+add2.size());
    //TODO! one of the following could in theroy be ommited 
    thrust::fill(timestamps_res.begin(),timestamps_res.end(),-1);
    thrust::fill(values_res.begin(),values_res.end(),-1);

    thrust::unique_by_key(merged_timestamps.begin(),merged_values.begin(),values_res.begin());
        for (int i=0 ; i < merged_timestamps.size();i++){
      std::cout << "unq[" << i << "] = " <<merged_timestamps[i]<<" | "<<merged_values[i] << std::endl;
    }
    int invalids = thrust::count(values_res.begin(),values_res.end(),-1);
    
    thrust::copy(merged_timestamps.begin(), merged_timestamps.end()-invalids, 
                  result_timestamps+invalids);
    thrust::copy(values_res.begin(), values_res.end()-invalids, 
              result_values+invalids);

    for (int i=0 ; i < result->size;i++){
      std::cout << "Final[" << i << "] = " <<result_values[i]<< std::endl;
    }
    
    
    exit(1);

}