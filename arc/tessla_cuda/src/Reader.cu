#include "hip/hip_runtime.h"
#include "Reader.cuh"
#include <assert.h>
#include <fstream>
#include <iostream>
#include <regex>
#include <string>
#include <vector>
#include <time.h>
using namespace std;

#define NEW_READER

Reader::Reader(string inputFile) {
    this->FILENAME = inputFile;
#ifdef NEW_READER
    readStreams();
#endif
}

IntInStream::IntInStream() {
}

UnitInStream::UnitInStream() {
}

void printArray(int* array, size_t len, string name) {
    printf("%s : [", name.c_str());
    for (int i=0; i < len - 1; i++) {
        printf("%d, ", array[i]);
    }
    printf("%d]\n", array[len-1]);
}

void Reader::readStreams() {
    fstream file;
    clock_t start = clock();
    file.open(this->FILENAME, ios::in);
    printf("read file %s\n", this->FILENAME.c_str());
    if (file.is_open())  {
        string buf;
        int i = 0;
        while (getline(file, buf)) {
            //printf("LINE %d\n", i);
            i++;
            buf.erase(std::remove_if(buf.begin(), buf.end(),::isspace), buf.end());
            size_t colPos = buf.find(':');
            size_t eqPos = buf.find('=');
            if (colPos == std::string::npos || eqPos == std::string::npos) {
                char buff[50];
                std::snprintf(buff, sizeof(buff), "Line %d: invalid pattern", i);
                throw std::runtime_error(buff);
            }
            int timestamp = stoi(buf, nullptr);
            string name = buf.substr(colPos+1, eqPos-colPos-1);

            size_t post_eq = eqPos + 1;
            try {
                int value = stoi(buf.substr(post_eq));

                // check if exists in map
                if (this->intStreams.find(name) == this->intStreams.end()) {
                    //printf("Create int stream %s\n", name.c_str());
                    shared_ptr<IntInStream> s = make_shared<IntInStream>();
                    this->intStreams.insert(std::pair<string,shared_ptr<IntInStream>>(name, s));
                }

                if (this->intStreams.find(name) != this->intStreams.end()) {
                    //printf("Insert (%d, %d) int stream %s\n", timestamp, value, name.c_str());
                    this->intStreams.find(name)->second->timestamps.push_back(timestamp);
                    this->intStreams.find(name)->second->values.push_back(value);
                } else {
                    throw std::runtime_error("Error in IntStream map insertion for Stream \"" + name + "\"");
                }

            } catch (std::invalid_argument &ia) {
                // check unit event validity
                if (buf.substr(post_eq) != "()") {
                    throw std::runtime_error("Invalid string \"" + buf.substr(post_eq) +
                                             "\" at RHS of non-int stream");
                }

                // check if exists in map
                if (this->unitStreams.find(name) == this->unitStreams.end()) {
                    //printf("Create unit stream %s\n", name.c_str());
                    shared_ptr<UnitInStream> s = make_shared<UnitInStream>();
                    this->unitStreams.insert(std::pair<string,shared_ptr<UnitInStream>>(name, s));
                }

                if (this->unitStreams.find(name) != this->unitStreams.end()) {
                    //printf("Insert %d in unit stream %s\n", timestamp, name.c_str());
                    this->unitStreams.find(name)->second->timestamps.push_back(timestamp);
                    //printf("last elem in %s: %d\n", name.c_str(), this->unitStreams.find(name)->second->timestamps.back());
                    //printf("Post insert unit stream %s\n", name.c_str());
                } else {
                    throw std::runtime_error("Error in UnitStream map insertion for Stream \"" + name + "\"");
                }
            }
        }
    }
    clock_t dur = clock() - start;
    printf("READING TOOK %ld us\n", dur*1000000/CLOCKS_PER_SEC);
}

#ifndef NEW_READER
UnitStream Reader::getUnitStream(string name) {
    fstream file;
    file.open(this->FILENAME, ios::in);
    vector<int> timestamps;

    if (file.is_open()) {
        string buf;
        regex pattern("([0-9]+):\\s*([A-Za-z][0-9A-Za-z]*)\\s*=\\s*\\(\\)\\s*");
        while (getline(file, buf)) {
            // match each line to regex
            smatch matches;
            if (regex_match(buf, matches, pattern)) {
                if (name.compare(matches[2]) == 0) {
                    int timestamp = stoi(matches[1]);
                    timestamps.push_back(timestamp);
                }
            }
        }
        file.close();
    }

    size_t timestampsCnt = timestamps.size();
    if (timestampsCnt == 0) {
        cout << "Warning: Stream " << name << " is not present in the input file but was read!" << "\n";
        //exit(1);
    }

    int *timestampsA = (int*) malloc(timestampsCnt * sizeof(int));
    memset(timestampsA, 0, timestampsCnt * sizeof(int));
    copy(timestamps.begin(), timestamps.end(), timestampsA);

    /*
    printf("%s: size=%d\n", name.c_str(), timestampsCnt);
    if (timestampsCnt < 10000) {
        printArray(timestampsA, timestampsCnt, "ts (" + name + ")");
    }
     */

    UnitStream readStream = UnitStream(timestampsA, timestampsCnt);
    return readStream;
}

IntStream Reader::getIntStream(string name) {
    fstream file;
    file.open(this->FILENAME, ios::in);
    vector<int> timestamps;
    vector<int> values;

    printf("read file %s\n", this->FILENAME.c_str());
    if (file.is_open()) {
        string buf;
        // match each line to regex
        regex pattern("([0-9]+):\\s*([A-Za-z][0-9A-Za-z]*)\\s*=\\s*(-?[0-9]+)\\s*");
        while (getline(file, buf)) {
            // match each line to regex
            smatch matches;
            if (regex_match(buf, matches, pattern)) {
                if (name.compare(matches[2]) == 0) {
                    int timestamp = stoi(matches[1]);
                    int value = stoi(matches[3]);
                    timestamps.push_back(timestamp);
                    values.push_back(value);
                }
            }
        }
        file.close();
    }

    assert(timestamps.size() == values.size());
    size_t timestampsCnt = timestamps.size();
    if (timestampsCnt == 0) {
        cout << "Warning: Stream " << name << " is not present in the input file but was read!" << "\n";
        //exit(1);
    }

    size_t mallocSize = timestampsCnt * sizeof(int);
    int *timestampsA = (int*) malloc(mallocSize);
    int *valuesA = (int*) malloc(mallocSize);
    memset(timestampsA, 0, mallocSize);
    memset(valuesA, 0, mallocSize);
    copy(timestamps.begin(), timestamps.end(), timestampsA);
    copy(values.begin(), values.end(), valuesA);

    /*
    printf("%s: size=%d\n", name.c_str(), timestampsCnt);
    if (timestampsCnt < 10000) {
        printArray(timestampsA, timestampsCnt, "ts (" + name + ")");
        printArray(valuesA, timestampsCnt, "vs (" + name + ")");
    }
    */

    IntStream readStream = IntStream(timestampsA, valuesA, timestampsCnt);
    return readStream;
}
#endif

#ifdef NEW_READER
UnitStream Reader::getUnitStream(string name) {
    if (this->unitStreams.find(name) != this->unitStreams.end()) {
        vector<int> *timestamps = &this->unitStreams.find(name)->second->timestamps;
        size_t mallocSize = timestamps->size() * sizeof(int);
        size_t size = timestamps->size();
        int *timestampsA = (int*) malloc(mallocSize);
        copy(timestamps->begin(), timestamps->end(), timestampsA);
        /*
        printf("%s: size=%zu\n", name.c_str(), timestamps->size());
        if (timestamps->size() < 10000) {
            printArray(&(*timestamps)[0], timestamps->size(), "ts (" + name + ")");
        }
         */
        return {timestampsA, size};
    } else {
        throw std::runtime_error("could not find unit stream \"" + std::string(name) + "\"");
    }
}

IntStream Reader::getIntStream(string name) {
    if (this->intStreams.find(name) != this->intStreams.end()) {
        vector<int> *timestamps = &this->intStreams.find(name)->second->timestamps;
        vector<int> *values = &this->intStreams.find(name)->second->values;
        size_t mallocSize = timestamps->size() * sizeof(int);
        size_t size = timestamps->size();
        int *timestampsA = (int*) malloc(mallocSize);
        int *valuesA = (int*) malloc(mallocSize);
        clock_t start = clock();
        copy(timestamps->begin(), timestamps->end(), timestampsA);
        copy(values->begin(), values->end(), valuesA);
        clock_t time = clock() - start;
        printf("MEMCPY TIME USED:: %ld\n", time*1000000/CLOCKS_PER_SEC);
        /*
        printf("%s: size=%zu\n", name.c_str(), size);
        if (size < 10000) {
            printArray(&(*timestamps)[0], timestamps->size(), "ts (" + name + ")");
            printArray(&(*values)[0], values->size(), "vs (" + name + ")");
        }
         */
        return {timestampsA, valuesA, size};
    } else {
        throw std::runtime_error("could not find int stream \"" + std::string(name) + "\"");
    }
}
#endif